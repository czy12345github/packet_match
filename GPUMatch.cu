#include "hip/hip_runtime.h"
#include "GPUMatch.h"

#include <fstream>
#include <sstream>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

__device__ bool is_equal(uint8_t *data1, uint8_t *data2, uint16_t len){
  for(uint16_t i = 0; i < len; ++i){
    if(data1[i] != data2[i]) return false;
  }
  return true;
}

__device__ int data_match(uint8_t *data, uint16_t len, struct Rule *rules, int numOfRules){
  struct Rule *rule;
  int n;
  for(int i = 0; i < numOfRules; ++i){
    rule = rules + i;
    if(len < rule->size) continue;
    n = len - rule->size + 1;
    for(int j = 0; j < n; ++j){
      if(is_equal(rule->content, data+j, rule->size)) return i;
    }
  }
  return -1;
}

__global__ void gpu_filter_packets(struct Packet *pkts, struct Rule *rules, int numOfRules, int *result){
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int start = index * pktsPerThread;
  for(int i = start; i < start + pktsPerThread; ++i){
    result[i] = data_match(pkts[i].data, pkts[i].pkt_len, rules, numOfRules);
  }
}

GPUMatch::GPUMatch(std::string ruleFile):
force_quit(false), array_num(0), has_data(false){
  hipMalloc(&gpu_pkts, ArraySize * sizeof(struct Packet));
  hipMalloc(&gpu_match_result, ArraySize * sizeof(int));

  cpu_match_result = new int[ArraySize];

  read_rules(ruleFile);
}

GPUMatch::~GPUMatch(){
  hipFree(gpu_pkts);

  delete []cpu_rules;
  hipFree(gpu_rules);

  delete []cpu_match_result;
  hipFree(gpu_match_result);
}

void GPUMatch::read_rules(std::string ruleFile){
  std::ifstream ifs(ruleFile);

  std::vector<std::vector<int>> rules;

  std::string line;

  int x;
  while(getline(ifs, line)){
    std::istringstream iss(line);
    std::vector<int> data;
    iss >> std::hex;
    while(iss >> x) data.push_back(x);
    rules.push_back(data);
  }
  ifs.close();

  int N = rules.size();
  cpu_rules = new Rule[N];
  for(int i = 0; i < N; ++i){
    cpu_rules[i].size = rules[i].size();
    for(int j = 0; j < rules[i].size(); ++j){
      cpu_rules[i].content[j] = (uint8_t)(0xff & rules[i][j]);
    }
  }
  numOfRules = N;

  hipMalloc(&gpu_rules, N * sizeof(struct Rule));
  hipMemcpy(gpu_rules, cpu_rules, N * sizeof(struct Rule), hipMemcpyHostToDevice);
}

void GPUMatch::process(){
  struct Packet *pkts;

  while(!force_quit){

    if(has_data){
      q_m.lock();
      pkts = q.front();
      q.pop();
      --array_num;
      if(array_num == 0) has_data = false;
      q_m.unlock();

      process_pkts(pkts);
    }
  }
}

void GPUMatch::process_pkts(struct Packet *pkts){
  hipMemcpy(gpu_pkts, pkts, ArraySize * sizeof(struct Packet), hipMemcpyHostToDevice);

  gpu_filter_packets<<<numBlocks, threadsPerBlock>>>(gpu_pkts, gpu_rules, numOfRules, gpu_match_result);
  hipDeviceSynchronize();

  hipMemcpy(cpu_match_result, gpu_match_result, ArraySize * sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < ArraySize; ++i){
    if(cpu_match_result[i] >= 0){
      std::cout << pkts[i].src_ip << " " << pkts[i].dst_ip << " " << cpu_match_result[i] << std::endl;
    }
  }

  free(pkts);
}
